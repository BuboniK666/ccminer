#include "hip/hip_runtime.h"
extern "C" {
//#include <sph/sph_blake.h>
#include "lyra2/Lyra2.h"
}

#include <miner.h>
#include <cuda_helper.h>
//#include <sph/blake2s.h>
#include <sph/sph_types.h>

extern void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, bool gtx750ti);

//extern void blake2s_setBlock(const uint32_t* input, const uint32_t ptarget7);
//extern void blake2s_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce);
//extern void blake2s_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint32_t ptarget7);

// Blake2s

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

#include "cuda_helper.h"

#ifdef __CUDA_ARCH__

__device__ __forceinline__
uint32_t ROR8(const uint32_t a) {
	return __byte_perm(a, 0, 0x0321);
}

__device__ __forceinline__
uint32_t ROL16(const uint32_t a) {
	return __byte_perm(a, 0, 0x1032);
}

#else
#define ROR8(u)  (u >> 8)
#define ROL16(u) (u << 16)
#endif

__device__ __forceinline__
uint32_t xor3x(uint32_t a, uint32_t b, uint32_t c)
{
	uint32_t result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result) : "r"(a), "r"(b), "r"(c)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result = a^b^c;
#endif
	return result;
}

static const uint32_t blake2s_IV[8] = {
	0x6A09E667UL, 0xBB67AE85UL, 0x3C6EF372UL, 0xA54FF53AUL,
	0x510E527FUL, 0x9B05688CUL, 0x1F83D9ABUL, 0x5BE0CD19UL
};


#define TPB 1024
#define NPT 256
#define maxResults 16
#define NBN 1
__constant__ uint32_t _ALIGN(32) midstate[20];

static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

static const uint8_t blake2s_sigma[10][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};

#define G(r,i,a,b,c,d) \
	do { \
		a = a + b + m[blake2s_sigma[r][2*i+0]]; \
		d = SPH_ROTR32(d ^ a, 16); \
		c = c + d; \
		b = SPH_ROTR32(b ^ c, 12); \
		a = a + b + m[blake2s_sigma[r][2*i+1]]; \
		d = SPH_ROTR32(d ^ a, 8); \
		c = c + d; \
		b = SPH_ROTR32(b ^ c, 7); \
			} while(0)
#define ROUND(r)  \
	do { \
		G(r,0,v[0],v[4],v[ 8],v[12]); \
		G(r,1,v[1],v[5],v[ 9],v[13]); \
		G(r,2,v[2],v[6],v[10],v[14]); \
		G(r,3,v[3],v[7],v[11],v[15]); \
		G(r,4,v[0],v[5],v[10],v[15]); \
		G(r,5,v[1],v[6],v[11],v[12]); \
		G(r,6,v[2],v[7],v[ 8],v[13]); \
		G(r,7,v[3],v[4],v[ 9],v[14]); \
			} while(0)

#define GS4(a,b,c,d,e,f,a1,b1,c1,d1,e1,f1,a2,b2,c2,d2,e2,f2,a3,b3,c3,d3,e3,f3){ \
	a += b + e;		a1+= b1 + e1;	 	a2+= b2 + e2;		a3+= b3 + e3; \
	d  = ROL16( d ^ a);	d1 = ROL16(d1 ^ a1);	d2 = ROL16(d2 ^ a2);	d3 = ROL16(d3 ^ a3); \
	c +=d; 			c1+=d1;			c2+=d2;			c3+=d3;\
	b  = ROTR32(b ^ c, 12); b1 = ROTR32(b1^c1, 12);	b2 = ROTR32(b2^c2, 12);	b3 = ROTR32(b3^c3, 12); \
	a += b + f;		a1+= b1 + f1;		a2+= b2 + f2;		a3+= b3 + f3; \
	d  = ROR8(d ^ a);	d1 = ROR8(d1^a1);	d2 = ROR8(d2^a2);	d3 = ROR8(d3^a3); \
	c  += d;		c1 += d1;		c2 += d2;		c3 += d3;\
	b  = ROTR32(b ^ c, 7);	b1 = ROTR32(b1^c1, 7);	b2 = ROTR32(b2^c2, 7);	b3 = ROTR32(b3^c3, 7); \
			}


static void allium_blake2s_setBlock(const uint32_t* input, const uint32_t ptarget7)
{
	uint32_t _ALIGN(64) m[16];
	uint32_t _ALIGN(64) v[16];
	uint32_t _ALIGN(64) h[21];

	//	COMPRESS
	for (int i = 0; i < 16; ++i)
		m[i] = input[i];

	h[0] = 0x01010020 ^ blake2s_IV[0];
	h[1] = blake2s_IV[1];
	h[2] = blake2s_IV[2]; h[3] = blake2s_IV[3];
	h[4] = blake2s_IV[4]; h[5] = blake2s_IV[5];
	h[6] = blake2s_IV[6]; h[7] = blake2s_IV[7];

	for (int i = 0; i < 8; ++i)
		v[i] = h[i];

	v[8] = blake2s_IV[0];		v[9] = blake2s_IV[1];
	v[10] = blake2s_IV[2];		v[11] = blake2s_IV[3];
	v[12] = 64 ^ blake2s_IV[4];	v[13] = blake2s_IV[5];
	v[14] = blake2s_IV[6];		v[15] = blake2s_IV[7];

	ROUND(0); ROUND(1);
	ROUND(2); ROUND(3);
	ROUND(4); ROUND(5);
	ROUND(6); ROUND(7);
	ROUND(8); ROUND(9);

	for (int i = 0; i < 8; ++i)
		h[i] ^= v[i] ^ v[i + 8];

	h[16] = input[16];
	h[17] = input[17];
	h[18] = input[18];

	h[8] = 0x6A09E667; h[9] = 0xBB67AE85;
	h[10] = 0x3C6EF372; h[11] = 0xA54FF53A;
	h[12] = 0x510E522F; h[13] = 0x9B05688C;
	h[14] = ~0x1F83D9AB; h[15] = 0x5BE0CD19;

	h[0] += h[4] + h[16];
	h[12] = SPH_ROTR32(h[12] ^ h[0], 16);
	h[8] += h[12];
	h[4] = SPH_ROTR32(h[4] ^ h[8], 12);
	h[0] += h[4] + h[17];
	h[12] = SPH_ROTR32(h[12] ^ h[0], 8);
	h[8] += h[12];
	h[4] = SPH_ROTR32(h[4] ^ h[8], 7);

	h[1] += h[5] + h[18];
	h[13] = SPH_ROTR32(h[13] ^ h[1], 16);
	h[9] += h[13];
	h[5] = ROTR32(h[5] ^ h[9], 12);

	h[2] += h[6];
	h[14] = SPH_ROTR32(h[14] ^ h[2], 16);
	h[10] += h[14];
	h[6] = SPH_ROTR32(h[6] ^ h[10], 12);
	h[2] += h[6];
	h[14] = SPH_ROTR32(h[14] ^ h[2], 8);
	h[10] += h[14];
	h[6] = SPH_ROTR32(h[6] ^ h[10], 7);

	h[19] = h[7]; //constant h[7] for nonce check

	h[3] += h[7];
	h[15] = SPH_ROTR32(h[15] ^ h[3], 16);
	h[11] += h[15];
	h[7] = SPH_ROTR32(h[7] ^ h[11], 12);
	h[3] += h[7];
	h[15] = SPH_ROTR32(h[15] ^ h[3], 8);
	h[11] += h[15];
	h[7] = SPH_ROTR32(h[7] ^ h[11], 7);

	h[1] += h[5];
	h[3] += h[4];
	h[14] = SPH_ROTR32(h[14] ^ h[3], 16);

	h[2] += h[7];
	if (ptarget7 == 0){
		h[19] = SPH_ROTL32(h[19], 7); //align the rotation with v[7] v[15];
	}
	hipMemcpyToSymbol(HIP_SYMBOL(midstate), h, 20 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

__global__ __launch_bounds__(TPB, 1)
void allium_blake2s_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce)
{
	const uint32_t step = gridDim.x * blockDim.x;

	uint32_t m[3];
	uint32_t v[16];

	m[0] = midstate[16];
	m[1] = midstate[17];
	m[2] = midstate[18];

	const uint32_t h7 = midstate[19];

	for (uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x; thread <threads; thread += step)
	{
#pragma unroll
		for (int i = 0; i<16; i++){
			v[i] = midstate[i];
		}

		uint32_t nonce = cuda_swab32(startNonce + thread);

		//		Round( 0 );
		v[1] += nonce;
		v[13] = ROR8(v[13] ^ v[1]);
		v[9] += v[13];
		v[5] = ROTR32(v[5] ^ v[9], 7);

		v[1] += v[6];
		v[0] += v[5];

		v[13] = ROL16(v[13] ^ v[2]);		v[12] = ROL16(v[12] ^ v[1]);		v[15] = ROL16(v[15] ^ v[0]);

		v[8] += v[13];				v[11] += v[12];				v[9] += v[14];				v[10] += v[15];
		v[7] = ROTR32(v[7] ^ v[8], 12);	v[6] = ROTR32(v[6] ^ v[11], 12);	v[4] = ROTR32(v[4] ^ v[9], 12);	v[5] = ROTR32(v[5] ^ v[10], 12);
		v[2] += v[7];				v[1] += v[6];				v[3] += v[4];				v[0] += v[5];
		v[13] = ROR8(v[13] ^ v[2]);		v[12] = ROR8(v[12] ^ v[1]);		v[14] = ROR8(v[14] ^ v[3]);		v[15] = ROR8(v[15] ^ v[0]);
		v[8] += v[13];				v[11] += v[12];				v[9] += v[14];				v[10] += v[15];
		v[6] = ROTR32(v[6] ^ v[11], 7);	v[7] = ROTR32(v[7] ^ v[8], 7);	v[4] = ROTR32(v[4] ^ v[9], 7);	v[5] = ROTR32(v[5] ^ v[10], 7);

		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], 0, 0, v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], m[1], 0, v[1], v[6], v[11], v[12], m[0], m[2], v[2], v[7], v[8], v[13], 0, 0, v[3], v[4], v[9], v[14], 0, nonce);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], 0, m[0], v[2], v[6], v[10], v[14], 0, m[2], v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], 0, 0, v[1], v[6], v[11], v[12], nonce, 0, v[2], v[7], v[8], v[13], 0, m[1], v[3], v[4], v[9], v[14], 0, 0);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], nonce, m[1], v[2], v[6], v[10], v[14], 0, 0, v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], m[2], 0, v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], 0, m[0], v[3], v[4], v[9], v[14], 0, 0);
		GS4(v[0], v[4], v[8], v[12], 0, m[0], v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], m[2], 0, v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], 0, m[1], v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], 0, 0, v[3], v[4], v[9], v[14], nonce, 0);
		GS4(v[0], v[4], v[8], v[12], m[2], 0, v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], m[0], 0, v[3], v[7], v[11], v[15], 0, nonce);
		GS4(v[0], v[5], v[10], v[15], 0, 0, v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], 0, 0, v[3], v[4], v[9], v[14], m[1], 0);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], m[1], 0, v[2], v[6], v[10], v[14], 0, 0, v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], m[0], 0, v[1], v[6], v[11], v[12], 0, nonce, v[2], v[7], v[8], v[13], 0, m[2], v[3], v[4], v[9], v[14], 0, 0);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], 0, m[1], v[3], v[7], v[11], v[15], nonce, 0);
		GS4(v[0], v[5], v[10], v[15], 0, m[0], v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], 0, 0, v[3], v[4], v[9], v[14], m[2], 0);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], 0, nonce, v[3], v[7], v[11], v[15], m[0], 0);
		GS4(v[0], v[5], v[10], v[15], 0, m[2], v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], m[1], 0, v[3], v[4], v[9], v[14], 0, 0);
		GS4(v[0], v[4], v[8], v[12], 0, m[2], v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], 0, 0, v[3], v[7], v[11], v[15], m[1], 0);

		v[0] += v[5];
		v[2] += v[7] + nonce;
		v[15] = ROL16(v[15] ^ v[0]);
		v[13] = ROL16(v[13] ^ v[2]);
		v[10] += v[15];
		v[8] += v[13];
		v[5] = ROTR32(v[5] ^ v[10], 12);
		v[7] = ROTR32(v[7] ^ v[8], 12);
		v[0] += v[5];
		v[2] += v[7];
		v[15] = ROTR32(v[15] ^ v[0], 1);
		v[13] = ROR8(v[13] ^ v[2]);

		v[8] += v[13];

		if (xor3x(v[7], h7, v[8]) == v[15]){
			uint32_t pos = atomicInc(&resNonce[0], 0xffffffff) + 1;
			if (pos < maxResults)
				resNonce[pos] = nonce;
			return;
		}
	}
}

__global__ __launch_bounds__(TPB, 1)
void allium_blake2s_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint32_t ptarget7)
{
	const uint32_t step = gridDim.x * blockDim.x;

	uint32_t m[3];
	uint32_t v[16];

	m[0] = midstate[16];
	m[1] = midstate[17];
	m[2] = midstate[18];

	const uint32_t h7 = midstate[19];

	for (uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x; thread <threads; thread += step){
#pragma unroll
		for (int i = 0; i<16; i++){
			v[i] = midstate[i];
		}

		uint32_t nonce = cuda_swab32(startNonce + thread);
		//		Round( 0 );
		v[1] += nonce;
		v[13] = ROR8(v[13] ^ v[1]);
		v[9] += v[13];
		v[5] = ROTR32(v[5] ^ v[9], 7);

		v[1] += v[6];
		v[0] += v[5];

		v[12] = ROL16(v[12] ^ v[1]);
		v[13] = ROL16(v[13] ^ v[2]);
		v[15] = ROL16(v[15] ^ v[0]);

		v[11] += v[12];				v[8] += v[13];				v[9] += v[14];				v[10] += v[15];
		v[6] = ROTR32(v[6] ^ v[11], 12);	v[7] = ROTR32(v[7] ^ v[8], 12);	v[4] = ROTR32(v[4] ^ v[9], 12);	v[5] = ROTR32(v[5] ^ v[10], 12);
		v[1] += v[6];				v[2] += v[7];				v[3] += v[4];				v[0] += v[5];
		v[12] = ROR8(v[12] ^ v[1]);		v[13] = ROR8(v[13] ^ v[2]);		v[14] = ROR8(v[14] ^ v[3]);		v[15] = ROR8(v[15] ^ v[0]);
		v[11] += v[12]; 				v[8] += v[13];				v[9] += v[14];				v[10] += v[15];
		v[6] = ROTR32(v[6] ^ v[11], 7);	v[7] = ROTR32(v[7] ^ v[8], 7);	v[4] = ROTR32(v[4] ^ v[9], 7);	v[5] = ROTR32(v[5] ^ v[10], 7);

		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], 0, 0, v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], m[1], 0, v[1], v[6], v[11], v[12], m[0], m[2], v[2], v[7], v[8], v[13], 0, 0, v[3], v[4], v[9], v[14], 0, nonce);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], 0, m[0], v[2], v[6], v[10], v[14], 0, m[2], v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], 0, 0, v[1], v[6], v[11], v[12], nonce, 0, v[2], v[7], v[8], v[13], 0, m[1], v[3], v[4], v[9], v[14], 0, 0);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], nonce, m[1], v[2], v[6], v[10], v[14], 0, 0, v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], m[2], 0, v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], 0, m[0], v[3], v[4], v[9], v[14], 0, 0);
		GS4(v[0], v[4], v[8], v[12], 0, m[0], v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], m[2], 0, v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], 0, m[1], v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], 0, 0, v[3], v[4], v[9], v[14], nonce, 0);
		GS4(v[0], v[4], v[8], v[12], m[2], 0, v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], m[0], 0, v[3], v[7], v[11], v[15], 0, nonce);
		GS4(v[0], v[5], v[10], v[15], 0, 0, v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], 0, 0, v[3], v[4], v[9], v[14], m[1], 0);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], m[1], 0, v[2], v[6], v[10], v[14], 0, 0, v[3], v[7], v[11], v[15], 0, 0);
		GS4(v[0], v[5], v[10], v[15], m[0], 0, v[1], v[6], v[11], v[12], 0, nonce, v[2], v[7], v[8], v[13], 0, m[2], v[3], v[4], v[9], v[14], 0, 0);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], 0, m[1], v[3], v[7], v[11], v[15], nonce, 0);
		GS4(v[0], v[5], v[10], v[15], 0, m[0], v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], 0, 0, v[3], v[4], v[9], v[14], m[2], 0);
		GS4(v[0], v[4], v[8], v[12], 0, 0, v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], 0, nonce, v[3], v[7], v[11], v[15], m[0], 0);
		GS4(v[0], v[5], v[10], v[15], 0, m[2], v[1], v[6], v[11], v[12], 0, 0, v[2], v[7], v[8], v[13], m[1], 0, v[3], v[4], v[9], v[14], 0, 0);
		GS4(v[0], v[4], v[8], v[12], 0, m[2], v[1], v[5], v[9], v[13], 0, 0, v[2], v[6], v[10], v[14], 0, 0, v[3], v[7], v[11], v[15], m[1], 0);

		//		GS(9,4,v[ 0],v[ 5],v[10],v[15]);
		v[0] += v[5];
		v[2] += v[7] + nonce;
		v[15] = ROL16(v[15] ^ v[0]);
		v[13] = ROL16(v[13] ^ v[2]);
		v[10] += v[15];
		v[8] += v[13];
		v[5] = ROTR32(v[5] ^ v[10], 12);
		v[7] = ROTR32(v[7] ^ v[8], 12);
		v[0] += v[5];
		v[2] += v[7];
		v[15] = ROR8(v[15] ^ v[0]);
		v[13] = ROR8(v[13] ^ v[2]);

		v[8] += v[13];
		v[7] = ROTR32(v[7] ^ v[8], 7);

		if (xor3x(h7, v[7], v[15]) <= ptarget7){
			uint32_t pos = atomicInc(&resNonce[0], 0xffffffff) + 1;
			if (pos < maxResults)
				resNonce[pos] = nonce;
			return;
		}
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// ALLIUM
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

static uint64_t* d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern "C" void allium_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hashA[8], hashB[8];

	blake2s_hash(hashA, input);
	LYRA2(hashB, 32, hashA, 32, hashA, 32, 1, 8, 8);

	memcpy(state, hashB, 32);
}

static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;
static __thread bool gtx750ti = false;

extern "C" int scanhash_allium(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t _ALIGN(64) endiandata[20];
	const uint32_t Htarg = ptarget[7];
	const uint32_t first_nonce = pdata[19];
	uint32_t nonce = first_nonce;

	int dev_id = device_map[thr_id];
	int rc = 0;

	// blake2s
	uint32_t *resNonces;

	if (opt_benchmark)
		ptarget[7] = 0x006fff;

	const dim3 grid((throughput + (NPT*TPB) - 1) / (NPT*TPB));
	const dim3 block(TPB);

	static __thread bool gtx750ti;
	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		cuda_get_arch(thr_id);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], maxResults * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], maxResults * sizeof(uint32_t)), -1);

		int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 17 : 16;
		if (device_sm[dev_id] <= 500) intensity = 15;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);
		gtx750ti = (strstr(props.name, "750 Ti") != NULL);

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		if (device_sm[dev_id] >= 500)
		{
			size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
			CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
			lyra2_cpu_init(thr_id, throughput, d_matrix[thr_id]);
		}

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}
	resNonces = h_resNonce[thr_id];

	for (int k = 0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);
	allium_blake2s_setBlock(endiandata, ptarget[7]);

	uint32_t _ALIGN(64) hash[8];
	do {
		be32enc(&endiandata[19], nonce);

		if (ptarget[7]) {
			allium_blake2s_gpu_hash_nonce<<<grid, block>>>(throughput, nonce, d_resNonce[thr_id], ptarget[7]);
		}
		else {
			allium_blake2s_gpu_hash_nonce<<<grid, block>>>(throughput, nonce, d_resNonce[thr_id]);
		}

		
		be32enc(&d_hash[thr_id], (uint32_t) d_resNonce[thr_id]);
		//d_hash[thr_id] = (uint32_t)d_resNonce[thr_id];

		lyra2_cpu_hash_32(thr_id, throughput, nonce, d_hash[thr_id], gtx750ti);

		hipMemcpy(resNonces, d_hash[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

		if (resNonces[0])
		{
			hipMemcpy(resNonces, d_hash[thr_id], maxResults*sizeof(uint32_t), hipMemcpyDeviceToHost);
			hipMemset(d_hash[thr_id], 0x00, sizeof(uint32_t));

			if (resNonces[0] >= maxResults) {
				gpulog(LOG_WARNING, thr_id, "candidates flood: %u", resNonces[0]);
				resNonces[0] = maxResults - 1;
			}

			nonce = sph_bswap32(resNonces[1]);
			be32enc(&endiandata[19], nonce);
			allium_hash(hash, endiandata);

			if (hash[7] <= Htarg && fulltest(hash, ptarget)) {
				gpulog(LOG_INFO, thr_id, "Found valid nonce");
				work->nonces[0] = nonce;
				work->valid_nonces = 1;
				work_set_target_ratio(work, hash);
				pdata[19] = nonce;
				*hashes_done = pdata[19] - first_nonce;
				return work->valid_nonces;
			}
		}

		if (nonce + throughput > max_nonce) {
			nonce = max_nonce;
			break;
		}

		nonce += throughput;
	} while (!work_restart[thr_id].restart);

	pdata[19] = nonce;
	*hashes_done = pdata[19] - first_nonce + 1;

	return 0;
}

// cleanup
extern "C" void free_allium(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	if (device_sm[dev_id] >= 350)
		hipFree(d_matrix[thr_id]);
	//lyra2Z_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
